#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <fcns.h>

// Definition of Matrix Type
typedef struct {
    int width;
    int height;
    double* elements;
} Matrix;

// Definition of Matrix Filter Function CPU
void MatFilter(const Matrix myfilter, Matrix oldimage, Matrix newimage)
{
    // Load  to device memory
    Matrix d_myfilter;
    d_myfilter.width = myfilter.width; 
	d_myfilter.height = myfilter.height;
    size_t size = myfilter.width * myfilter.height * sizeof(double);
    hipMalloc(&d_myfilter.elements, size);
    hipMemcpy(d_myfilter.elements, myfilter.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_myimage;
    d_oldimage.width = oldimage.width; 
	d_oldimage.height = oldimage.height;
    size = oldimage.width * oldimage.height * sizeof(double);
    hipMalloc(&d_oldimage.elements, size);
    hipMemcpy(d_oldimage.elements, oldimage.elements, size,
               hipMemcpyHostToDevice);
	Matrix d_myimage;
    d_newimage.width = newimage.width;
    d_newimage.height = newimage.height;
    size = newimage.width * newimage.height * sizeof(double);
    hipMalloc(&d_newimage.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((newimage.width+BLOCK_SIZE-1) / dimBlock.x, 
		(newimage.height+BLOCK_SIZE-1)/ dimBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_filter, d_oldimage, d_newimage);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapseTime;
    hipEventElapsedTime(&elapseTime, start, stop);
    printf("Time to generate: %f ms\n", elapseTime);

    // Read C from device memory
    hipMemcpy(newimage.elements, d_newimage.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_filter.elements);
    hipFree(d_oldimage.elements);
    hipFree(d_newimage.elements);
}

// Definition of Matrix Filter Function GPU
__global__ void MatFilterKernel(Matrix filter, Matrix oldimage, Matrix newimage)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    double Cvalue = 0;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < A_HEIGHT && col < B_WIDTH)
    {
        for (int i = 0; i < A.width; ++i)
            Cvalue += A.elements[row * A.width + i]
                    * B.elements[i * B.width + col];
        C.elements[row * C.width + col] = Cvalue;
    }
}

